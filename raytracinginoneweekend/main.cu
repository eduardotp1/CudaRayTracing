#include "hip/hip_runtime.h"
//==================================================================================================
// Written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is distributed
// without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication along
// with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
// This code was change following the tutorial of Roger Allen https://devblogs.nvidia.com/accelerated-ray-tracing-cuda/
//==================================================================================================

#include <iostream>
#include "sphere.h"
#include "hitable_list.h"
#include "float.h"
#include "vec3.h"
#include "ray.h"
#include <chrono>

// traca os raios de luz
__device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f*(unit_direction.y() + 1.0f);
        return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

// hitable *random_scene() {
//     int n = 500;
//     hitable **list = new hitable*[n+1];
//     list[0] =  new sphere(vec3(0,-1000,0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
//     int i = 1;
//     for (int a = -11; a < 11; a++) {
//         for (int b = -11; b < 11; b++) {
//             float choose_mat = drand48();
//             vec3 center(a+0.9*drand48(),0.2,b+0.9*drand48()); 
//             if ((center-vec3(4,0.2,0)).length() > 0.9) { 
//                 if (choose_mat < 0.8) {  // diffuse
//                     list[i++] = new sphere(center, 0.2, new lambertian(vec3(drand48()*drand48(), drand48()*drand48(), drand48()*drand48())));
//                 }
//                 else if (choose_mat < 0.95) { // metal
//                     list[i++] = new sphere(center, 0.2,
//                             new metal(vec3(0.5*(1 + drand48()), 0.5*(1 + drand48()), 0.5*(1 + drand48())),  0.5*drand48()));
//                 }
//                 else {  // glass
//                     list[i++] = new sphere(center, 0.2, new dielectric(1.5));
//                 }
//             }
//         }
//     }

//     list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
//     list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
//     list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

//     return new hitable_list(list,i);
// }

// pinta a imagem
__global__ void rgb(vec3 *fb, int max_x, int max_y,vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin, hitable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r, world);
}

// instancia as esferas
__global__ void create_sphere(hitable **d_list, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *(d_list+2)   = new sphere(vec3(0,10,-1), 15);
        *(d_list+3) = new sphere(vec3(0,20,-1), 10);
        *(d_list+4)   = new sphere(vec3(0,2,-1), 3);
        *(d_list+5) = new sphere(vec3(0,30,-1), 2);
        *d_world    = new hitable_list(d_list,6);
    }
}
// deleta memorias
__global__ void free_memory(hitable **d_list, hitable **d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

int main() {
    int nx = 1200;
    int ny = 800;
    // int ns = 10;
    int tx = 8;//divisoes que vai ser cortada a imagem
    int ty = 8;//divisoes que vai ser cortada a imagem
    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);
    using namespace std::chrono;
    high_resolution_clock::time_point begin = high_resolution_clock::now();

    // allocate FB
    vec3 *fb;
    hipMallocManaged((void **)&fb, fb_size);
    hitable **d_list;
    hipMalloc((void **)&d_list, 6*sizeof(hitable *));
    hitable **d_world;
    hipMalloc((void **)&d_world, sizeof(hitable *));
    create_sphere<<<1,1>>>(d_list,d_world);
    hipDeviceSynchronize();

    dim3 block_size(nx/tx+1,ny/ty+1);//tamanho de cada grid
    dim3 size_grid(tx,ty);//tamanho do grid
    rgb<<<block_size, size_grid>>>(fb, nx, ny, vec3(-2.0, -1.0, -1.0), vec3(4.0, 0.0, 0.0), vec3(0.0, 2.0, 0.0), vec3(0.0, 0.0, 0.0),d_world);//manda para a GPU calcular
    hipDeviceSynchronize();

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    // hitable *list[5];

    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;

            // size_t pixel_index = j*3*nx + i*3;
            int ir = int(255.99*fb[pixel_index][0]);
            int ig = int(255.99*fb[pixel_index][1]);
            int ib = int(255.99*fb[pixel_index][2]);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    high_resolution_clock::time_point end = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(end - begin);
    std::cerr << "Tempo: " << time_span.count();

    hipDeviceSynchronize();
    free_memory<<<1,1>>>(d_list,d_world);
    hipFree(d_list);
    hipFree(d_world);
    hipFree(fb);
    hipDeviceReset();
}